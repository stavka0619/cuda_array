#include "hip/hip_runtime.h"
#include "hipArray.h"
#include "operators.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <boost/progress.hpp>
#include <hipblas.h>

using namespace std;
using namespace cuda_array;


// __global__ void setValue(float * dest, int nx, int ny)
// {
// 	const int tid = (blockIdx.y*1 + blockIdx.x)*blockDim.x + threadIdx.x;
//         if (tid < nx*ny)
//         {
//             float value = dest[tid]*2+0.5;
//             dest[tid] = value;
//         }
// }

// __global__ void setValue(cuda_array::cuArray<float,2> a)
// {
// 	const int tid = (blockIdx.y*1 + blockIdx.x)*blockDim.y*threadIdx.x + threadIdx.y;
//     if (tid < a.numElements())
//         {
//             float value = tid;
//             a(threadIdx.x, threadIdx.y) = value;
//         }
// }

int main()
{
    int nx=512;
    int nz=120;
    cuArray<float,3> a(nx,nx,nz);
    cuArray<float,3> b(nx,nx,nz);
    cuArray<float,3> c(nx,nx,nz);
    const int sz = nx*nx*nz;
    
    const int N = 100;
    
    
    float* aa = new float[sz];
    float* bb = new float[sz];
    for (int i=0;i<sz;i++)
    {
        aa[i]=i;
        bb[i]=i;
    }
    a.copyfromHost(aa);
    b.copyfromHost(aa);
    c.copyfromHost(aa);
    // setValue<<<grid,threads>>>(a);
    // setValue<<<grid,threads>>>(b);
    {
        boost::progress_timer timer;
        for (int ii=0;ii<N;ii++)
            for (int jj=0;jj<sz;jj++)
                bb[jj] += aa[jj];
    }
    
    {
        boost::progress_timer timer;
        for (int ii=0;ii<N;ii++)
            c += a;
        hipDeviceSynchronize();
    }

    hipblasHandle_t blas_handle;
    hipblasCreate(&blas_handle);
    float one = 1;
    {
        boost::progress_timer timer;
        for (int ii=0;ii<N;ii++)
        {
            hipblasSaxpy(blas_handle, b.size(), &one, a.data(), 1, b.data(), 1); // update f=f+s
        }
            hipDeviceSynchronize();
    }
    cout<<"calculate completed"<<endl;
    c.copytoHost(bb);
    for (int i=0;i<10;i++)
    {
        for (int j=0;j<10;j++)
            cout<<bb[i*10+j]<<' ';
        cout<<endl;
    }

    delete [] aa;
    delete [] bb;
    
    return 0;
}
