#include "hip/hip_runtime.h"
#include "hipArray.h"
#include "hip/hip_runtime.h"
#include <iostream>
using namespace std;
using namespace cuda_array;


// __global__ void setValue(float * dest, int nx, int ny)
// {
// 	const int tid = (blockIdx.y*1 + blockIdx.x)*blockDim.x + threadIdx.x;
//         if (tid < nx*ny)
//         {
//             float value = dest[tid]*2+0.5;
//             dest[tid] = value;
//         }
// }

__global__ void setValue(cuda_array::cuArray<float,2> a)
{
	const int tid = (blockIdx.y*1 + blockIdx.x)*blockDim.y*threadIdx.x + threadIdx.y;
    if (tid < a.numElements())
        {
            float value = tid;
            a(threadIdx.x, threadIdx.y) = value;
        }
}

int main()
{
    dim3 grid(1,1);
    dim3 threads(10,10,1);
    cuda_array::cuArray<float,2> a(3,3);
     cuda_array::cuArray<float,1> b(3);
     cuda_array::cuArray<float,1> c(3);
   float aa[100];
    for (int i=0;i<100;i++)
        aa[i]=i;
    a.copyfromHost(aa);
//    cuda_array::cuArray<float,2> b(a,Range::all(),Range(3,7));
    setValue<<<grid,threads>>>(a);
    setValue<<<grid,threads>>>(b);
    c = a*b;
    
    
    hipDeviceSynchronize();
    float bb[100];
    
      
    a.copytoHost(bb);
    
    for (int i=0;i<10;i++)
    {
        for (int j=0;j<10;j++)
            cout<<bb[i*10+j]<<' ';
        cout<<endl;
    }
    return 0;
}
